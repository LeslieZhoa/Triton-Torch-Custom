#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static __host__ __device__ __forceinline__ int64_t floor_div(int64_t a, int64_t b) {
  int64_t c = a / b;

  if (c * b > a) {
    c--;
  }

  return c;
}

struct UpFirDn2DKernelParams {
  int64_t up_x;
  int64_t up_y;
  int64_t down_x;
  int64_t down_y;
  int64_t pad_x0;
  int64_t pad_x1;
  int64_t pad_y0;
  int64_t pad_y1;

  int64_t major_dim;
  int64_t in_h;
  int64_t in_w;
  int64_t minor_dim;
  int64_t kernel_h;
  int64_t kernel_w;
  int64_t out_h;
  int64_t out_w;
  int64_t loop_major;
  int64_t loop_x;
};

template <typename scalar_t>
__global__ void upfirdn2d_kernel_large(scalar_t *out, const scalar_t *input,
                                       const scalar_t *kernel,
                                       const UpFirDn2DKernelParams p) {
  int64_t minor_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t out_y = minor_idx / p.minor_dim;
  minor_idx -= out_y * p.minor_dim;
  int64_t out_x_base = blockIdx.y * p.loop_x * blockDim.y + threadIdx.y;
  int64_t major_idx_base = blockIdx.z * p.loop_major;

  if (out_x_base >= p.out_w || out_y >= p.out_h ||
      major_idx_base >= p.major_dim) {
    return;
  }
  int v_zero = 0;
  int64_t mid_y = out_y * p.down_y + p.up_y - 1 - p.pad_y0;
  int64_t in_y = min(max(int(floor_div(mid_y, p.up_y)), v_zero), int(p.in_h));
  int64_t h = min(max(int(floor_div(mid_y + p.kernel_h, p.up_y)), v_zero), int(p.in_h)) - in_y;
  int64_t kernel_y = mid_y + p.kernel_h - (in_y + 1) * p.up_y;
  int64_t zzero = 0;
  for (int64_t loop_major = 0, major_idx = major_idx_base;
       loop_major < p.loop_major && major_idx < p.major_dim;
       loop_major++, major_idx++) {
    for (int64_t loop_x = 0, out_x = out_x_base;
         loop_x < p.loop_x && out_x < p.out_w; loop_x++, out_x += blockDim.y) {
      int64_t mid_x = out_x * p.down_x + p.up_x - 1 - p.pad_x0;
      int64_t in_x = min(max(int(floor_div(mid_x, p.up_x)), v_zero), int(p.in_w));
      int64_t w = min(max(floor_div(mid_x + p.kernel_w, p.up_x),zzero), p.in_w) - in_x;
      int64_t kernel_x = mid_x + p.kernel_w - (in_x + 1) * p.up_x;

      const scalar_t *x_p =
          &input[((major_idx * p.in_h + in_y) * p.in_w + in_x) * p.minor_dim +
                 minor_idx];
      const scalar_t *k_p = &kernel[kernel_y * p.kernel_w + kernel_x];
      int64_t x_px = p.minor_dim;
      int64_t k_px = -p.up_x;
      int64_t x_py = p.in_w * p.minor_dim;
      int64_t k_py = -p.up_y * p.kernel_w;

      scalar_t v = 0.0f;

      for (int64_t y = 0; y < h; y++) {
        for (int64_t x = 0; x < w; x++) {
          v += static_cast<scalar_t>(*x_p) * static_cast<scalar_t>(*k_p);
          x_p += x_px;
          k_p += k_px;
        }

        x_p += x_py - w * x_px;
        k_p += k_py - w * k_px;
      }

      out[((major_idx * p.out_h + out_y) * p.out_w + out_x) * p.minor_dim +
          minor_idx] = v;
    }
  }
}

template <typename scalar_t, int64_t up_x, int64_t up_y, int64_t down_x, int64_t down_y,
          int64_t kernel_h, int64_t kernel_w, int64_t tile_out_h, int64_t tile_out_w>
__global__ void upfirdn2d_kernel(scalar_t *out, const scalar_t *input,
                                 const scalar_t *kernel,
                                 const UpFirDn2DKernelParams p) {
  const int64_t tile_in_h = ((tile_out_h - 1) * down_y + kernel_h - 1) / up_y + 1;
  const int64_t tile_in_w = ((tile_out_w - 1) * down_x + kernel_w - 1) / up_x + 1;

  __shared__ volatile float sk[kernel_h][kernel_w];
  __shared__ volatile float sx[tile_in_h][tile_in_w];

  int64_t minor_idx = blockIdx.x;
  int64_t tile_out_y = minor_idx / p.minor_dim;
  minor_idx -= tile_out_y * p.minor_dim;
  tile_out_y *= tile_out_h;
  int64_t tile_out_x_base = blockIdx.y * p.loop_x * tile_out_w;
  int64_t major_idx_base = blockIdx.z * p.loop_major;

  if (tile_out_x_base >= p.out_w | tile_out_y >= p.out_h |
      major_idx_base >= p.major_dim) {
    return;
  }

  for (int64_t tap_idx = threadIdx.x; tap_idx < kernel_h * kernel_w;
       tap_idx += blockDim.x) {
    int64_t ky = tap_idx / kernel_w;
    int64_t kx = tap_idx - ky * kernel_w;
    scalar_t v = 0.0;

    if (kx < p.kernel_w & ky < p.kernel_h) {
      v = kernel[(p.kernel_h - 1 - ky) * p.kernel_w + (p.kernel_w - 1 - kx)];
    }

    sk[ky][kx] = v;
  }

  for (int64_t loop_major = 0, major_idx = major_idx_base;
       loop_major < p.loop_major & major_idx < p.major_dim;
       loop_major++, major_idx++) {
    for (int64_t loop_x = 0, tile_out_x = tile_out_x_base;
         loop_x < p.loop_x & tile_out_x < p.out_w;
         loop_x++, tile_out_x += tile_out_w) {
      int64_t tile_mid_x = tile_out_x * down_x + up_x - 1 - p.pad_x0;
      int64_t tile_mid_y = tile_out_y * down_y + up_y - 1 - p.pad_y0;
      int64_t tile_in_x = floor_div(tile_mid_x, up_x);
      int64_t tile_in_y = floor_div(tile_mid_y, up_y);

      __syncthreads();

      for (int64_t in_idx = threadIdx.x; in_idx < tile_in_h * tile_in_w;
           in_idx += blockDim.x) {
        int64_t rel_in_y = in_idx / tile_in_w;
        int64_t rel_in_x = in_idx - rel_in_y * tile_in_w;
        int64_t in_x = rel_in_x + tile_in_x;
        int64_t in_y = rel_in_y + tile_in_y;

        scalar_t v = 0.0;

        if (in_x >= 0 & in_y >= 0 & in_x < p.in_w & in_y < p.in_h) {
          v = input[((major_idx * p.in_h + in_y) * p.in_w + in_x) *
                        p.minor_dim +
                    minor_idx];
        }

        sx[rel_in_y][rel_in_x] = v;
      }

      __syncthreads();
      for (int64_t out_idx = threadIdx.x; out_idx < tile_out_h * tile_out_w;
           out_idx += blockDim.x) {
        int64_t rel_out_y = out_idx / tile_out_w;
        int64_t rel_out_x = out_idx - rel_out_y * tile_out_w;
        int64_t out_x = rel_out_x + tile_out_x;
        int64_t out_y = rel_out_y + tile_out_y;

        int64_t mid_x = tile_mid_x + rel_out_x * down_x;
        int64_t mid_y = tile_mid_y + rel_out_y * down_y;
        int64_t in_x = floor_div(mid_x, up_x);
        int64_t in_y = floor_div(mid_y, up_y);
        int64_t rel_in_x = in_x - tile_in_x;
        int64_t rel_in_y = in_y - tile_in_y;
        int64_t kernel_x = (in_x + 1) * up_x - mid_x - 1;
        int64_t kernel_y = (in_y + 1) * up_y - mid_y - 1;

        scalar_t v = 0.0;

#pragma unroll
        for (int64_t y = 0; y < kernel_h / up_y; y++)
#pragma unroll
          for (int64_t x = 0; x < kernel_w / up_x; x++)
            v += sx[rel_in_y + y][rel_in_x + x] *
                 sk[kernel_y + y * up_y][kernel_x + x * up_x];

        if (out_x < p.out_w & out_y < p.out_h) {
          out[((major_idx * p.out_h + out_y) * p.out_w + out_x) * p.minor_dim +
              minor_idx] = v;
        }
      }
    }
  }
}

torch::Tensor upfirdn2d_op(const torch::Tensor &input,
                           const torch::Tensor &kernel, int64_t up_x, int64_t up_y,
                           int64_t down_x, int64_t down_y, int64_t pad_x0, int64_t pad_x1,
                           int64_t pad_y0, int64_t pad_y1) {
  int curDevice = -1;
  hipGetDevice(&curDevice);
  hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

  UpFirDn2DKernelParams p;

  auto x = input.contiguous();
  auto k = kernel.contiguous();

  p.major_dim = x.size(0);
  p.in_h = x.size(1);
  p.in_w = x.size(2);
  p.minor_dim = x.size(3);
  p.kernel_h = k.size(0);
  p.kernel_w = k.size(1);
  p.up_x = up_x;
  p.up_y = up_y;
  p.down_x = down_x;
  p.down_y = down_y;
  p.pad_x0 = pad_x0;
  p.pad_x1 = pad_x1;
  p.pad_y0 = pad_y0;
  p.pad_y1 = pad_y1;

  p.out_h = (p.in_h * p.up_y + p.pad_y0 + p.pad_y1 - p.kernel_h + p.down_y) /
            p.down_y;
  p.out_w = (p.in_w * p.up_x + p.pad_x0 + p.pad_x1 - p.kernel_w + p.down_x) /
            p.down_x;

  auto out =
      at::empty({p.major_dim, p.out_h, p.out_w, p.minor_dim}, x.options());

  int64_t mode = -1;

  int64_t tile_out_h = -1;
  int64_t tile_out_w = -1;

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 1;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 3 && p.kernel_w <= 3) {
    mode = 2;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 2 && p.up_y == 2 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 3;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 2 && p.up_y == 2 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 2 && p.kernel_w <= 2) {
    mode = 4;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 2 && p.down_y == 2 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 5;
    tile_out_h = 8;
    tile_out_w = 32;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 2 && p.down_y == 2 &&
      p.kernel_h <= 2 && p.kernel_w <= 2) {
    mode = 6;
    tile_out_h = 8;
    tile_out_w = 32;
  }

  dim3 block_size;
  dim3 grid_size;

  if (tile_out_h > 0 && tile_out_w > 0) {
    p.loop_major = (p.major_dim - 1) / 16384 + 1;
    p.loop_x = 1;
    block_size = dim3(32 * 8, 1, 1);
    grid_size = dim3(((p.out_h - 1) / tile_out_h + 1) * p.minor_dim,
                     (p.out_w - 1) / (p.loop_x * tile_out_w) + 1,
                     (p.major_dim - 1) / p.loop_major + 1);
  } else {
    p.loop_major = (p.major_dim - 1) / 16384 + 1;
    p.loop_x = 4;
    block_size = dim3(4, 32, 1);
    grid_size = dim3((p.out_h * p.minor_dim - 1) / block_size.x + 1,
                     (p.out_w - 1) / (p.loop_x * block_size.y) + 1,
                     (p.major_dim - 1) / p.loop_major + 1);
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "upfirdn2d_cuda", [&] {
    switch (mode) {
    case 1:
      upfirdn2d_kernel<scalar_t, 1, 1, 1, 1, 4, 4, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 2:
      upfirdn2d_kernel<scalar_t, 1, 1, 1, 1, 3, 3, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 3:
      upfirdn2d_kernel<scalar_t, 2, 2, 1, 1, 4, 4, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 4:
      upfirdn2d_kernel<scalar_t, 2, 2, 1, 1, 2, 2, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 5:
      upfirdn2d_kernel<scalar_t, 1, 1, 2, 2, 4, 4, 8, 32>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 6:
      upfirdn2d_kernel<scalar_t, 1, 1, 2, 2, 4, 4, 8, 32>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    default:
      upfirdn2d_kernel_large<scalar_t><<<grid_size, block_size, 0, stream>>>(
          out.data_ptr<scalar_t>(), x.data_ptr<scalar_t>(),
          k.data_ptr<scalar_t>(), p);
    }
  });

  return out;
}
